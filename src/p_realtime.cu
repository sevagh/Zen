#include "hpss.h"
#include "nppdefs.h"
#include "nppi.h"
#include "rhythm_toolkit/hpss.h"
#include <cuda/hip/hip_runtime.h>
#include <cuda/hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <math.h>
#include <stdio.h>
#include <thrust/complex.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/fill.h>
#include <thrust/functional.h>
#include <thrust/host_vector.h>
#include <thrust/replace.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>

// real hpss code is below
// the public namespace is to hide cuda details away from the public interface
rhythm_toolkit::hpss::PRealtimeGPU::PRealtimeGPU(float fs,
                                 std::size_t hop,
                                 float beta,
                                 rhythm_toolkit::io::IOGPU& io)
	: io(io)
{
	p_impl = new rhythm_toolkit_private::hpss::PRealtimeGPU(fs, hop, beta);
}

rhythm_toolkit::hpss::PRealtimeGPU::PRealtimeGPU(float fs,
                                 std::size_t hop,
                                 rhythm_toolkit::io::IOGPU& io)
	: PRealtimeGPU(fs, hop, 2.5, io) {};

// best-performing defaults
rhythm_toolkit::hpss::PRealtimeGPU::PRealtimeGPU(float fs, rhythm_toolkit::io::IOGPU& io)
	: PRealtimeGPU(fs, 256, 2.5, io) {};

void rhythm_toolkit::hpss::PRealtimeGPU::process_next_hop()
{
	p_impl->process_next_hop(io.device_in);
	thrust::copy(p_impl->percussive_out.begin(), p_impl->percussive_out.end(), io.device_out);
}

rhythm_toolkit::hpss::PRealtimeGPU::~PRealtimeGPU() {
	delete p_impl;
}

void rhythm_toolkit_private::hpss::PRealtimeGPU::process_next_hop(thrust::device_ptr<float> in_hop)
{
	// following the previous iteration
	// we rotate the percussive and harmonic arrays to get them ready
	// for the next hop and next overlap add
	thrust::copy(percussive_out.begin() + hop, percussive_out.end(),
	             percussive_out.begin());
	thrust::fill(percussive_out.begin() + hop, percussive_out.end(), 0.0);

	// append latest hop samples e.g. input = input[hop:] + current_hop
	thrust::copy(input.begin() + hop, input.end(), input.begin());
	thrust::copy(in_hop, in_hop + hop, input.begin() + hop);

	// populate curr_fft with input .* square root von hann window
	thrust::transform(input.begin(), input.end(), win.window.begin(),
	                  curr_fft.begin(), rhythm_toolkit_private::hpss::window_functor());

	// zero out the second half of the fft
	thrust::fill(curr_fft.begin() + nwin, curr_fft.end(),
	             thrust::complex<float>{0.0, 0.0});
	hipfftExecC2C(plan_forward, fft_ptr, fft_ptr, HIPFFT_FORWARD);

	// rotate stft matrix to move the oldest column to the end
	// copy curr_fft into the last column of the stft
	thrust::copy(
	    sliding_stft.begin() + nfft, sliding_stft.end(), sliding_stft.begin());
	thrust::copy(curr_fft.begin(), curr_fft.end(), sliding_stft.end() - nfft);

	// calculate the magnitude of the stft
	thrust::transform(sliding_stft.begin(), sliding_stft.end(), s_mag.begin(),
	                  rhythm_toolkit_private::hpss::complex_abs_functor());

	// apply median filter in horizontal and vertical directions with NPP
	// to create percussive and harmonic spectra
	nppiFilterMedian_32f_C1R(thrust::raw_pointer_cast(s_mag.data()), nstep,
	                         thrust::raw_pointer_cast(harmonic_matrix.data()),
	                         nstep, medfilt_roi, harmonic_filter_mask,
	                         harmonic_anchor, harmonic_buffer);

	nppiFilterMedian_32f_C1R(
	    thrust::raw_pointer_cast(s_mag.data()), nstep,
	    thrust::raw_pointer_cast(percussive_matrix.data()), nstep, medfilt_roi,
	    percussive_filter_mask, percussive_anchor, percussive_buffer);

	// compute percussive mask from harmonic + percussive magnitude spectra
	// the last column of percussive_matrix contains the mask to be applied to the initial fft
	thrust::transform(percussive_matrix.end() - nfft, percussive_matrix.end(),
	                  harmonic_matrix.end() - nfft, percussive_matrix.begin(),
	                  rhythm_toolkit_private::hpss::mask_functor(beta));

	// apply last column of percussive mask to recover percussive audio from original fft
	thrust::transform(curr_fft.begin(), curr_fft.end(),
	                  percussive_matrix.end()-nfft, curr_fft.begin(),
	                  rhythm_toolkit_private::hpss::apply_mask_functor());
	
	hipfftExecC2C(plan_backward, fft_ptr, fft_ptr, HIPFFT_BACKWARD);

	// now curr_fft has the current iteration's fresh samples
	// we overlap-add it the real part to the previous
	thrust::transform(curr_fft.begin(), curr_fft.begin() + nwin,
	                  percussive_out.begin(), percussive_out.begin(),
	                  rhythm_toolkit_private::hpss::overlap_add_functor(COLA_factor));
}
