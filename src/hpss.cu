#include <stdio.h>
#include <cuda/hip/hip_runtime.h>
#include <cuda/hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <thrust/sequence.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/replace.h>
#include <thrust/functional.h>
#include <thrust/complex.h>
#include <iostream>
#include "rhythm_toolkit_priv.h"

void cuda_hpss(thrust::host_vector<float>& x_, thrust::host_vector<float>& p_)
{
	thrust::device_vector<float> x(x_);
	thrust::device_vector<thrust::complex<float>> fft(x.size()/2 + 1);

	hipfftHandle plan_forward;
	hipfftHandle plan_backward;

	hipfftPlan1d(&plan_forward, x.size(), HIPFFT_R2C, 1);
	hipfftPlan1d(&plan_backward, x.size(), HIPFFT_C2R, 1);

	hipfftReal* x_cuda = (hipfftReal*)thrust::raw_pointer_cast(x.data());
	hipFloatComplex* fft_cuda = (hipFloatComplex*)thrust::raw_pointer_cast(fft.data());

	hipfftExecR2C(plan_forward, x_cuda, fft_cuda);
	hipfftExecC2R(plan_backward, fft_cuda, x_cuda);

	p_ = x;
}
