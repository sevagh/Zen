#include "hip/hip_runtime.h"
#include "util.h"
#include <cmath>
#include <float.h>
#include <fstream>
#include <iostream>
#include <iterator>
#include <map>
#include <sstream>
#include <string>
#include <vector>

std::vector<double>
test_util::sinewave(size_t size, double frequency, int sample_rate)
{
	size_t lut_size = size / 4;

	std::vector<int> lut{};
	double* _tone_single_channel = ( double* )malloc(sizeof(double) * size / 2);

	double doublef = ( double )frequency;
	double delta_phi = doublef * lut_size * 1.0 / sample_rate;
	double phase = 0.0;

	for (int i = 0; i < signed(lut_size); ++i) {
		lut.push_back(( int )roundf(0x7FFF * sinf(2.0 * M_PI * i / lut_size)));
	}

	double min = DBL_MAX;
	double max = -DBL_MAX;
	for (int i = 0; i < signed(size / 2); ++i) {
		int val = double(lut[( int )phase]);
		if (val > max) {
			max = val;
		}
		if (val < min) {
			min = val;
		}
		_tone_single_channel[i] = val;
		phase += delta_phi;
		if (phase >= lut_size)
			phase -= lut_size;
	}

	std::vector<double> tone_single_channel(
	    _tone_single_channel, _tone_single_channel + size / 2);

	return tone_single_channel;
}

std::vector<double> test_util::vec_from_file(std::string path)
{
	std::vector<double> data;
	std::ifstream infile(path);

	if (infile.fail()) {
		std::cerr << "File '" << path << "' doesn't exist, exiting"
		          << std::endl;
		exit(1);
	}

	double val;
	while (infile >> val)
		data.push_back(val);

	return data;
}
